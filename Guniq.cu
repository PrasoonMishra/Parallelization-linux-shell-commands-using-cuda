#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#include<vector>
#include<math.h>
#include<cstring>
#include<utility>
#include<iostream> 
#include<algorithm>
using namespace std;

//structure for holding the file data(lines of string)
struct fileData{
    int *equal;
    char **lines;
};

void findingArgument(char *str, int len, bool argumentCapture[], bool err[])
{
	for(int i=1;i<len;i++){
		switch(str[i]){
			case 'c': argumentCapture[0] = true; break;
			case 'd': argumentCapture[1] = true; break;
			case 'D': argumentCapture[2] = true; break;
			case 'u': argumentCapture[3] = true; break;
			case 'i': argumentCapture[4] = true; break;
			default:  err[0] = true;
		}
		if(err[0]) return;
	}
}

void printVersion(){
	cout<<"Guniq (version: 1.0) is a GPU based implementation of uniq linux command utility."
               "\nWritten by Prasoon Mishra with love | 2021" <<endl;
}

void printHelping(){
    cout<<"Usage: ./Guniq [OPTION]... [INPUT [OUTPUT]]"
          "\nFilter adjacent matching lines from INPUT,"
          "\nwriting to OUTPUT (or standard output)."
          "\n"
          "\nWith no options, matching lines are merged to the first occurrence."
          "\n"
          "\nArguments::"
          "\n-c     prefix lines by the number of occurrences"
          "\n-d     only print duplicate lines, one for each group"
          "\n-D     print all duplicate lines"
          "\n-u     only print unique lines"
          "\n-i     ignore differences in case when comparing"
          "\n--version     output version information and exit"
          "\n--help     display this help and exit" 
          "\n\nNote: 'Guniq' does not detect repeated lines unless they are adjacent."<<endl;
}

__global__ void caseSensitiveKernel(int *GPUequal, char *GPUlines, int length, int count_lines){
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if(id < count_lines-1){
        int fpos,spos,f;
        fpos = id*length;
        spos = (id+1)*length;
        f = 0;

        while(GPUlines[fpos] != '\n' && GPUlines[spos] != '\n'){
            if(GPUlines[fpos] != GPUlines[spos]){
                f = 1; break;
            }
            else{
                fpos++; spos++;
            } 
        }
        
        if(f != 1 && GPUlines[fpos] == GPUlines[spos]){
            GPUequal[id+1] = 1;
        }
    }
}

__global__ void caseInsensitiveKernel(int *GPUequal, char *GPUlines, int length, int count_lines){
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if(id < count_lines-1){
        int fpos,spos,f;
        fpos = id*length;
        spos = (id+1)*length;
        f = 0;

        while( GPUlines[fpos] != '\n' && GPUlines[spos] != '\n' ){
            if( GPUlines[fpos] >= 'A' && GPUlines[fpos] <= 'Z' ){
                if(GPUlines[fpos] != GPUlines[spos] && GPUlines[fpos] +32 != GPUlines[spos]){
                    f = 1; break;
                }
                else{
                    fpos++; spos++;        
                }
            }
            else if( GPUlines[fpos] >= 'a' && GPUlines[fpos] <= 'z' ){
                if(GPUlines[fpos] != GPUlines[spos] && GPUlines[fpos] -32 != GPUlines[spos]){
                    f = 1; break;
                }
                else{
                    fpos++; spos++;        
                }
            }
            else if(GPUlines[fpos] != GPUlines[spos]){
                f = 1; break;
            }
            else{
                fpos++; spos++;
            } 
        }//end of while
        
        if(f != 1 && GPUlines[fpos] == GPUlines[spos]){
            GPUequal[id+1] = 1;
        }
    }
}

void printError(int i){
        switch(i){
        case 0: cout<<"Error: Synatax not followed properly.\n i.e No arguments should be after file names."<<endl; break;
        case 1: cout<<"Error: Synatax not followed properly.\n i.e Not more than two file names(input & output) should be passed."<<endl; break;
        case 2: cout<<"Error: Synatax not followed properly.\n i.e Unknown arguments passed."<<endl; break;
        default: cout<<"Error: Synatax not followed properly. Use ./Guniq --help for further help."<<endl;; break;
    }
}

int main(int argc, char **argv){

	//error variable
	bool err[1];
	err[0] = false;

	//array for knowing which argument was used
	//argumentCapture capture info about ['c','d','D','u','i'] arguments by putting true/false at respective positions
	bool argumentCapture[5];
	for(int i=0; i<5; i++)
		argumentCapture[i] = false;

	//finding options used in this command
	bool temp = false;
	int fileNameIndex1 = -1, fileNameIndex2 = -1; 
	for(int i=1; i<argc; i++){

		if(strcmp(argv[i], "--version") == 0){
			printVersion();
			exit(0);
		}
		if(strcmp(argv[i], "--help") == 0){
			printHelping();
			exit(0);
		}

		if(strncmp(argv[i],"-", 1) == 0 && temp){
			printError(0);
			exit(0);
		}
		else if(strncmp(argv[i],"-", 1) == 0){
			int len = strlen(argv[i]);
			findingArgument(argv[i], len, argumentCapture, err);
		}
		else
		{
			temp = true;
			if(fileNameIndex1 == -1) fileNameIndex1 = i;
			else if(fileNameIndex2 == -1) fileNameIndex2 = i;
			else{
                printError(1);
				exit(0);
			}
		}

		if(err[0]){
			printError(2);
			exit(0);
		}
	}//end of for

	//Checking of above code
	// printf("c:%s\n", argumentCapture[0] ? "true" : "false");
	// printf("d:%s\n", argumentCapture[1] ? "true" : "false");
	// printf("D:%s\n", argumentCapture[2] ? "true" : "false");
	// printf("u:%s\n", argumentCapture[3] ? "true" : "false");
	// printf("i:%s\n", argumentCapture[4] ? "true" : "false");

	if(argumentCapture[0] && argumentCapture[2]){
		printf("Guniq: printing all duplicated lines and repeat counts is meaningless. Try './Guniq --help' for more information.\n");
		exit(0);
	}

	if(argumentCapture[1] && argumentCapture[3]){
		printf("Guniq: printing only duplicated lines only and printing only unique lines only is meaningless. Try './Guniq --help' for more information.\n");
		exit(0);
	}

	if(argumentCapture[1] && argumentCapture[2]){
		printf("Guniq: printing all duplicated lines and printing only one duplicate lines for each group is counter arguments. Hence meaningless. Try './Guniq --help' for more information.\n");
		exit(0);
	}

	if(argumentCapture[2] && argumentCapture[3]){
		printf("Guniq: printing only duplicated lines only and printing only unique lines only is meaningless. Try './Guniq --help' for more information.\n");
		exit(0);
	}

	if(fileNameIndex1 == -1 && fileNameIndex2 == -1){
		printf("Error: No file was passed as argument. Pls try again!\n");
		exit(0);
	}

	//Computing the uniq for arguments which do not have "i"
	if(!argumentCapture[4]){
		char *inputfilename = argv[fileNameIndex1];
        FILE *fileptr;
        fileptr = fopen(inputfilename , "r");

        if (fileptr == NULL){
            printf( "Error: Input file failed to open." );
            return 0;
        }

        // printf("%s\n",inputfilename);

        int count_lines = 0;
        int max_len = 0, max = 0;
        char chr;
        chr = getc(fileptr);
        while (chr != EOF)
        {
            //Count whenever new line is encountered
            if (chr == '\n'){
                //Calculating the total lines in string and also the max length of the string
                count_lines = count_lines + 1;
                if(max_len < max) max_len = max;
                max = 0;
            }
            else max++;

            //take next character from file.
            chr = getc(fileptr);
        }
        rewind(fileptr); 

        //variable declaration
        fileData data;
        int *GPUequal;
        char *GPUlines;
        char *datalines;

        //memory allocation
        data.equal = (int*) calloc(count_lines,sizeof(int));
        data.lines = (char **) malloc(count_lines * sizeof(char *));  
        for(int i=0; i<count_lines; i++){
            data.lines[i] = (char *) malloc((max_len+2) * sizeof(char));
        }
        datalines = (char *)malloc(count_lines * (max_len+2) * sizeof(char));
        hipMalloc(&GPUequal, count_lines * sizeof(int));
        hipMalloc(&GPUlines, count_lines * (max_len+2) * sizeof(char));      

        // printf("count_lines=%d, max_len=%d\n", count_lines, max_len);

        // char tempLine[max_len+1];
        int i=0;
        //copying the data
        char * line = NULL;
    	size_t len = 0;
    	ssize_t read;

    	while ((read = getline(&line, &len, fileptr)) != -1) {
    		strcpy(data.lines[i],line);
    		i++;
    	}
    	i=0; 

        fclose(fileptr);

        //Copying 2d array to 1d array in CPU
        int k = 0, j = 0;
        for( i=0 ; i<count_lines; i++ ){
            k = i*(max_len+2);
            while(data.lines[i][j] != '\n'){
                datalines[k] = data.lines[i][j];
                //cout<<datalines[k];
                k++;
                j++;
            }
            j = 0;
            datalines[k] = '\n';
            //cout<<datalines[k];
        }

        //For timing the cuda kernel, serial logic and print/file printing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliseconds = 0;
        hipEventRecord(start,0);


        //Initialization in GPU
        hipMemcpy(GPUequal, data.equal, count_lines * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(GPUlines, datalines, count_lines*(max_len+2)*sizeof(char), hipMemcpyHostToDevice);

        // for(i=0; i<count_lines; i++){
        // 	printf("%s",data.lines[i]);
        // }

        // processing of uniq main thing in GPU
        int StringLength = max_len+2;
        int NUM_THREADS = 1024;
        int NUM_BLOCKS = (ceil)((double)count_lines/NUM_THREADS);

        caseSensitiveKernel<<<NUM_BLOCKS, NUM_THREADS>>>(GPUequal, GPUlines, StringLength, count_lines);
        
        //copying back from GPU to CPU
        hipMemcpy(data.equal, GPUequal, count_lines * sizeof(int), hipMemcpyDeviceToHost);

        //Serial code        
        // for(i=0; i<count_lines-1; i++){
        //     if(strcmp( data.lines[i], data.lines[i+1]) == 0) data.equal[i+1] = 1;
        // }

        int backCounter = 0;
        for(i=count_lines-1; i>=0; i--){
            if(data.equal[i] == 1){
                data.equal[i] = -1;
                backCounter++;
            }
            else{
                data.equal[i] = backCounter;
                backCounter = 0;
            }
        }
        //serial code ends

        //printing
        if(argumentCapture[0]){
        	if(argumentCapture[1]){
	        	if(fileNameIndex2 != -1){
		            char *outputfilename = argv[fileNameIndex2];
		            fileptr = fopen(outputfilename , "w");
		            
		            if (fileptr == NULL){
		                printf( "Output file failed to open." );
		                exit(0);
	                }

	                for(i=0; i<count_lines; i++){
	                    if(data.equal[i] > 0){
	                        fprintf(fileptr, "%7d %s",data.equal[i]+1,data.lines[i]);
	                    }
	                }
	                fclose(fileptr);
        		}
        		else{
                	for(i=0; i<count_lines; i++){
                    	// printf("%d %s",data.equal[i], data.lines[i]);
                    	if(data.equal[i] > 0){
                        	printf("%7d %s",data.equal[i]+1, data.lines[i]);
                    	}
                	}     			  
        		}
        	}
        	else if(argumentCapture[3]){
	        	if(fileNameIndex2 != -1){
		            char *outputfilename = argv[fileNameIndex2];
		            fileptr = fopen(outputfilename , "w");
		            
		            if (fileptr == NULL){
		                printf( "Output file failed to open." );
		                exit(0);
	                }

	                for(i=0; i<count_lines; i++){
	                    if(data.equal[i] == 0){
	                        fprintf(fileptr, "%7d %s",data.equal[i]+1,data.lines[i]);
	                    }
	                }
	                fclose(fileptr);
        		}
        		else{
                	for(i=0; i<count_lines; i++){
                    	// printf("%d %s",data.equal[i], data.lines[i]);
                    	if(data.equal[i] == 0){
                        	printf("%7d %s",data.equal[i]+1, data.lines[i]);
                    	}
                	}       			  
        		}        		
        	}
        	else{
	        	if(fileNameIndex2 != -1){
		            char *outputfilename = argv[fileNameIndex2];
		            fileptr = fopen(outputfilename , "w");
		            
		            if (fileptr == NULL){
		                printf( "Output file failed to open." );
		                exit(0);
	                }

	                for(i=0; i<count_lines; i++){
	                    if(data.equal[i] >= 0){
	                        fprintf(fileptr, "%7d %s",data.equal[i]+1,data.lines[i]);
	                    }
	                }
	                fclose(fileptr);
        		}
        		else{
                	for(i=0; i<count_lines; i++){
                    	// printf("%d %s",data.equal[i], data.lines[i]);
                    	if(data.equal[i] >= 0){
                        	printf("%7d %s",data.equal[i]+1, data.lines[i]);
                    	}
                	}        			  
        		} 
        	}
        }
        else if(argumentCapture[1]){
        	if(fileNameIndex2 != -1){
	            char *outputfilename = argv[fileNameIndex2];
	            fileptr = fopen(outputfilename , "w");
	            
	            if (fileptr == NULL){
	                printf( "Output file failed to open." );
	                exit(0);
                }

                for(i=0; i<count_lines; i++){
                    if(data.equal[i] > 0){
                        fprintf(fileptr, "%s",data.lines[i]);
                    }
                }
                fclose(fileptr);
    		}
    		else{
            	for(i=0; i<count_lines; i++){
                	// printf("%d %s",data.equal[i], data.lines[i]);
                	if(data.equal[i] > 0){
                    	printf("%s",data.lines[i]);
                	}
            	}      			  
    		} 
        }
        else if(argumentCapture[2]){
	        	if(fileNameIndex2 != -1){
		            char *outputfilename = argv[fileNameIndex2];
		            fileptr = fopen(outputfilename , "w");
		            
		            if (fileptr == NULL){
		                printf( "Output file failed to open." );
		                exit(0);
	                }

	                for(i=0; i<count_lines; i++){
	                    if(data.equal[i] != 0){
	                        fprintf(fileptr, "%s",data.lines[i]);
	                    }
	                }
	                fclose(fileptr);
        		}
        		else{
                	for(i=0; i<count_lines; i++){
                    	// printf("%d %s",data.equal[i], data.lines[i]);
                    	if(data.equal[i] != 0){
                        	printf("%s",data.lines[i]);
                    	}
                	}      			  
        		} 
        }        	
        else if(argumentCapture[3]){
        	if(fileNameIndex2 != -1){
	            char *outputfilename = argv[fileNameIndex2];
	            fileptr = fopen(outputfilename , "w");
	            
	            if (fileptr == NULL){
	                printf( "Output file failed to open." );
	                exit(0);
                }

                for(i=0; i<count_lines; i++){
                    if(data.equal[i] == 0){
                        fprintf(fileptr, "%s",data.lines[i]);
                    }
                }
                fclose(fileptr);
    		}
    		else{
            	for(i=0; i<count_lines; i++){
                	// printf("%d %s",data.equal[i], data.lines[i]);
                	if(data.equal[i] == 0){
                    	printf("%s",data.lines[i]);
                	}
            	}        			  
    		} 
        }
        else{
        	if(fileNameIndex2 != -1){
	            char *outputfilename = argv[fileNameIndex2];
	            fileptr = fopen(outputfilename , "w");
	            
	            if (fileptr == NULL){
	                printf( "Output file failed to open." );
	                exit(0);
                }

                for(i=0; i<count_lines; i++){
                    if(data.equal[i] >= 0){
                        fprintf(fileptr, "%s",data.lines[i]);
                    }
                }
                fclose(fileptr);
    		}
    		else{
            	for(i=0; i<count_lines; i++){
                	// printf("%d %s",data.equal[i], data.lines[i]);
                	if(data.equal[i] >= 0){
                    	printf("%s",data.lines[i]);
                	}
            	}      			  
    		}         	
        }

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time taken by function to execute is: %.6f ms\n", milliseconds);


        //deleting memory allocated
        for(i = 0; i<count_lines; i++){
            delete(data.lines[i]);
        }
        delete(data.equal);
	}
	else{
        char *inputfilename = argv[fileNameIndex1];
        FILE *fileptr;
        fileptr = fopen(inputfilename , "r");

        if (fileptr == NULL){
            printf( "Error: Input file failed to open." );
            return 0;
        }

        // printf("%s\n",inputfilename);

        int count_lines = 0;
        int max_len = 0, max = 0;
        char chr;
        chr = getc(fileptr);
        while (chr != EOF)
        {
            //Count whenever new line is encountered
            if (chr == '\n'){
                //Calculating the total lines in string and also the max length of the string
                count_lines = count_lines + 1;
                if(max_len < max) max_len = max;
                max = 0;
            }
            else max++;

            //take next character from file.
            chr = getc(fileptr);
        }
        rewind(fileptr); 

        //variable declaration
        fileData data;
        int *GPUequal;
        char *GPUlines;
        char *datalines;

        //memory allocation
        data.equal = (int*) calloc(count_lines,sizeof(int));
        data.lines = (char **) malloc(count_lines * sizeof(char *));  
        for(int i=0; i<count_lines; i++){
            data.lines[i] = (char *) malloc((max_len+2) * sizeof(char));
        }
        datalines = (char *)malloc(count_lines * (max_len+2) * sizeof(char));
        hipMalloc(&GPUequal, count_lines * sizeof(int));
        hipMalloc(&GPUlines, count_lines * (max_len+2) * sizeof(char));      

        // printf("count_lines=%d, max_len=%d\n", count_lines, max_len);

        // char tempLine[max_len+1];
        int i=0;
        //copying the data
        char * line = NULL;
        size_t len = 0;
        ssize_t read;

        while ((read = getline(&line, &len, fileptr)) != -1) {
            strcpy(data.lines[i],line);
            i++;
        }
        i=0; 

        fclose(fileptr);

        //Copying 2d array to 1d array in CPU
        int k = 0, j = 0;
        for( i=0 ; i<count_lines; i++ ){
            k = i*(max_len+2);
            while(data.lines[i][j] != '\n'){
                datalines[k] = data.lines[i][j];
                //cout<<datalines[k];
                k++;
                j++;
            }
            j = 0;
            datalines[k] = '\n';
            //cout<<datalines[k];
        }

        //For timing the cuda kernel, serial logic and print/file printing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliseconds = 0;
        hipEventRecord(start,0);


        //Initialization in GPU
        hipMemcpy(GPUequal, data.equal, count_lines * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(GPUlines, datalines, count_lines*(max_len+2)*sizeof(char), hipMemcpyHostToDevice);

        // for(i=0; i<count_lines; i++){
        //  printf("%s",data.lines[i]);
        // }

        // processing of uniq main thing in GPU
        int StringLength = max_len+2;
        int NUM_THREADS = 1024;
        int NUM_BLOCKS = (ceil)((double)count_lines/NUM_THREADS);

        caseInsensitiveKernel<<<NUM_BLOCKS, NUM_THREADS>>>(GPUequal, GPUlines, StringLength, count_lines);
        
        //copying back from GPU to CPU
        hipMemcpy(data.equal, GPUequal, count_lines * sizeof(int), hipMemcpyDeviceToHost);

        //Serial code        
        // for(i=0; i<count_lines-1; i++){
        //     if(strcmp( data.lines[i], data.lines[i+1]) == 0) data.equal[i+1] = 1;
        // }

        int backCounter = 0;
        for(i=count_lines-1; i>=0; i--){
            if(data.equal[i] == 1){
                data.equal[i] = -1;
                backCounter++;
            }
            else{
                data.equal[i] = backCounter;
                backCounter = 0;
            }
        }
        //serial code ends

        //printing
        if(argumentCapture[0]){
            if(argumentCapture[1]){
                if(fileNameIndex2 != -1){
                    char *outputfilename = argv[fileNameIndex2];
                    fileptr = fopen(outputfilename , "w");
                    
                    if (fileptr == NULL){
                        printf( "Output file failed to open." );
                        exit(0);
                    }

                    for(i=0; i<count_lines; i++){
                        if(data.equal[i] > 0){
                            fprintf(fileptr, "%7d %s",data.equal[i]+1,data.lines[i]);
                        }
                    }
                    fclose(fileptr);
                }
                else{
                    for(i=0; i<count_lines; i++){
                        // printf("%d %s",data.equal[i], data.lines[i]);
                        if(data.equal[i] > 0){
                            printf("%7d %s",data.equal[i]+1, data.lines[i]);
                        }
                    }                 
                }
            }
            else if(argumentCapture[3]){
                if(fileNameIndex2 != -1){
                    char *outputfilename = argv[fileNameIndex2];
                    fileptr = fopen(outputfilename , "w");
                    
                    if (fileptr == NULL){
                        printf( "Output file failed to open." );
                        exit(0);
                    }

                    for(i=0; i<count_lines; i++){
                        if(data.equal[i] == 0){
                            fprintf(fileptr, "%7d %s",data.equal[i]+1,data.lines[i]);
                        }
                    }
                    fclose(fileptr);
                }
                else{
                    for(i=0; i<count_lines; i++){
                        // printf("%d %s",data.equal[i], data.lines[i]);
                        if(data.equal[i] == 0){
                            printf("%7d %s",data.equal[i]+1, data.lines[i]);
                        }
                    }                     
                }               
            }
            else{
                if(fileNameIndex2 != -1){
                    char *outputfilename = argv[fileNameIndex2];
                    fileptr = fopen(outputfilename , "w");
                    
                    if (fileptr == NULL){
                        printf( "Output file failed to open." );
                        exit(0);
                    }

                    for(i=0; i<count_lines; i++){
                        if(data.equal[i] >= 0){
                            fprintf(fileptr, "%7d %s",data.equal[i]+1,data.lines[i]);
                        }
                    }
                    fclose(fileptr);
                }
                else{
                    for(i=0; i<count_lines; i++){
                        // printf("%d %s",data.equal[i], data.lines[i]);
                        if(data.equal[i] >= 0){
                            printf("%7d %s",data.equal[i]+1, data.lines[i]);
                        }
                    }                     
                } 
            }
        }
        else if(argumentCapture[1]){
            if(fileNameIndex2 != -1){
                char *outputfilename = argv[fileNameIndex2];
                fileptr = fopen(outputfilename , "w");
                
                if (fileptr == NULL){
                    printf( "Output file failed to open." );
                    exit(0);
                }

                for(i=0; i<count_lines; i++){
                    if(data.equal[i] > 0){
                        fprintf(fileptr, "%s",data.lines[i]);
                    }
                }
                fclose(fileptr);
            }
            else{
                for(i=0; i<count_lines; i++){
                    // printf("%d %s",data.equal[i], data.lines[i]);
                    if(data.equal[i] > 0){
                        printf("%s",data.lines[i]);
                    }
                }                 
            } 
        }
        else if(argumentCapture[2]){
                if(fileNameIndex2 != -1){
                    char *outputfilename = argv[fileNameIndex2];
                    fileptr = fopen(outputfilename , "w");
                    
                    if (fileptr == NULL){
                        printf( "Output file failed to open." );
                        exit(0);
                    }

                    for(i=0; i<count_lines; i++){
                        if(data.equal[i] != 0){
                            fprintf(fileptr, "%s",data.lines[i]);
                        }
                    }
                    fclose(fileptr);
                }
                else{
                    for(i=0; i<count_lines; i++){
                        // printf("%d %s",data.equal[i], data.lines[i]);
                        if(data.equal[i] != 0){
                            printf("%s",data.lines[i]);
                        }
                    }                 
                } 
        }           
        else if(argumentCapture[3]){
            if(fileNameIndex2 != -1){
                char *outputfilename = argv[fileNameIndex2];
                fileptr = fopen(outputfilename , "w");
                
                if (fileptr == NULL){
                    printf( "Output file failed to open." );
                    exit(0);
                }

                for(i=0; i<count_lines; i++){
                    if(data.equal[i] == 0){
                        fprintf(fileptr, "%s",data.lines[i]);
                    }
                }
                fclose(fileptr);
            }
            else{
                for(i=0; i<count_lines; i++){
                    // printf("%d %s",data.equal[i], data.lines[i]);
                    if(data.equal[i] == 0){
                        printf("%s",data.lines[i]);
                    }
                }                     
            } 
        }
        else{
            if(fileNameIndex2 != -1){
                char *outputfilename = argv[fileNameIndex2];
                fileptr = fopen(outputfilename , "w");
                
                if (fileptr == NULL){
                    printf( "Output file failed to open." );
                    exit(0);
                }

                for(i=0; i<count_lines; i++){
                    if(data.equal[i] >= 0){
                        fprintf(fileptr, "%s",data.lines[i]);
                    }
                }
                fclose(fileptr);
            }
            else{
                for(i=0; i<count_lines; i++){
                    // printf("%d %s",data.equal[i], data.lines[i]);
                    if(data.equal[i] >= 0){
                        printf("%s",data.lines[i]);
                    }
                }                 
            }           
        }

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time taken by function to execute is: %.6f ms\n", milliseconds);


        //deleting memory allocated
        for(i = 0; i<count_lines; i++){
            delete(data.lines[i]);
        }
        delete(data.equal);
	}
}

